#include "hip/hip_runtime.h"
//STL includes
#include <iostream>
#include <vector>
#include <time.h>
#include <cmath>
#include <chrono>
#include <iomanip>

//Eigen includes
#include <Eigen/Dense>
#include <Eigen/Sparse>

//boost
#include "boost/program_options.hpp" 

//My own includes
#include "global_params.h"
#include "input_file_prep.h"
#include "BZ_CUDA_UTIL.h"
#include "base_layer.h"
#include "gpu_info_struct.h"
#include "custom_kernels.h"
#include "model.h"
#include "fileHelper.h"
#include "Eigen_Util.h"
#include "model.hpp"
#include "base_layer.hpp"
#include "LSTM.hpp"
#include "softmax.hpp"
#include "Input_To_Hidden_Layer.hpp"
#include "Hidden_To_Hidden_Layer.hpp"
#include "LSTM_HH.hpp"

//parse the command line from the user
void command_line_parse(global_params &params,int argc, char **argv) {
		
	//files for keeping the user input
	//if not s, 1st source, 2nd target, 3rd output weights name
	//if s, 1st target, 2nd output weights name
	std::vector<std::string> train_files;

	//files for force decoding
	//if not s, 1. source input file 2. target input file  3. neural network file name 4. output file name
	//if s, 1. target input file  2. neural network file name 3. output file name
	std::vector<std::string> test_files;

	//stuff for adaptive learning rate schedule
	//if not seq , 1st is source dev, 2nd is target dev
	//if seq 1st is target dev
	std::vector<std::string> adaptive_learning_rate;

	//lower and upper range for parameter initialization
	std::vector<precision> lower_upper_range;

	//for the kbest flag, 4 arguements must be entered for kbest, 1. number of best paths 2 input file name 
	//3. neural network file name (this is the output file you get after training the neural network)4. output file name
	std::vector<std::string> kbest_files;

	//for stoic gen, 1st neural network file, 2nd is output file name
	std::vector<std::string> stoicgen_files;

	//truncated softmax
	std::vector<std::string> trunc_info;

	//basic format setup
	namespace po = boost::program_options; 
	po::options_description desc("Options");
	desc.add_options() 
  		("help,h", "Run to get help on how to use the program")
  		("train,t",po::value<std::vector<std::string> > (&train_files)->multitoken(),"Train a model with input data file(s) and a name for the neural network output file"\
  			". \nFORMAT (if sequence to sequence): <source file name> <target file name> <neural network output name> "\
  			" \nFORMAT (if sequence): <target file name> <neural network output name>")
  		("force-decode,f",po::value<std::vector<std::string> > (&test_files)->multitoken(), "Get per line probability of dataset plus the perplexity\n"\
  			"FORMAT: (if sequence to sequence): <source file name> <target file name> <trained neural network file name> <output file name>\n"\
  			"FORMAT: (if sequence): <target file name> <trained neural network file name> <output file name>")
  		("stoch-gen,g", po::value<std::vector<std::string> > (&stoicgen_files)->multitoken(),"Do random generation for a sequence model, such as a language model\n"\
  			"FORMAT: <neural network file name> <output file name>")
  		("stoch-gen-len",po::value<int>(&params.sg_length) ,"How many tokens to let stoch-gen run for")
  		("sequence,s", "Train model that learns a sequence,such as language modeling. Default model is sequence to sequence model")
  		("learning-rate,l",po::value<precision>(&params.learning_rate),"Set learning rate")
  		("longest-sent,L",po::value<int>(&params.longest_sent),"Set the maximum sentence length for training.")
  		("hiddenstate-size,H",po::value<int>(&params.LSTM_size),"Set hiddenstate size")
  		("truncated-softmax,T",po::value<int>(&params.LSTM_size),"Set hiddenstate size\n"\
  			"FORMAT: <shortlist size> <sampled size>")
  		("source-vocab,v",po::value<int>(&params.source_vocab_size),"Set source vocab size")
  		("target-vocab,V",po::value<int>(&params.target_vocab_size),"Set target vocab size")
  		("shuffle",po::value<bool>(&params.shuffle),"true if you want to shuffle the train data")
  		("parameter-range,p",po::value<std::vector<precision> > (&lower_upper_range)->multitoken(),"parameter initialization range")
  		("number-epochs,n",po::value<int>(&params.num_epochs),"Set number of epochs")
  		("clip-gradients,c",po::value<precision>(&params.norm_clip),"Set gradient clipping threshold")
  		("adaptive-halve-lr",po::value<std::vector<std::string>> (&adaptive_learning_rate)->multitoken(),"Halve the learning rate"\
  			" when the perplexity on your specified dev set decreases from the previous half epoch\n"\
  			"FORMAT:")
  		("fixed-halve-lr",po::value<int> (&params.epoch_to_start_halving),"Halve the learning rate"\
  			" after a certain epoch, every half epoch afterwards by a specific amount")
  		("minibatch-size,m",po::value<int>(&params.minibatch_size),"Set minibatch size")
  		("kbest,k",po::value<std::vector<std::string> > (&kbest_files)->multitoken(),"Get k best paths in sequence to sequence model\n"\
  			"FORMAT: <how many paths> <source file name> <neural network file name> <output file name>") 
  		("beam-size,b",po::value<int>(&params.beam_size),"Set beam size for kbest paths")
  		("penalty,p",po::value<precision>(&params.penalty),"Set penalty for kbest decoding. The value entered"\
  			" will be added to the log probability score per target word decoded. This can make the model favor longer sentences for decoding");

    po::variables_map vm; 

	try {
		po::store(po::parse_command_line(argc, argv, desc), vm);
		po::notify(vm);
		
		//see if the user specified the help flag
		if ( vm.count("help") ) {

			std::cout << "\n------------------------------\n";
			std::cout << "This is Barret Zoph's CPU/GPU RNN library\n"
            << "The flags for the command line interface are below\n" 
            << "" << "\n";

			std::cout << desc << "\n";
			exit (EXIT_FAILURE);
		}

		//error checks to be sure only once of these options is set
		if (vm.count("train") && vm.count("kbest")) {
			std::cout << "ERROR: you cannot train and get kbest at the same time\n";
			exit (EXIT_FAILURE);
		}
		if (vm.count("train") && vm.count("force-decode")) {
			std::cout << "ERROR: you cannot train and force-decode at the same time\n";
			exit (EXIT_FAILURE);
		}
		if (vm.count("force-decode") && vm.count("kbest")) {
			std::cout << "ERROR: you cannot force-decode and get kbest at the same time\n";
			exit (EXIT_FAILURE);
		}
		if (!(vm.count("train") || vm.count("force-decode") || vm.count("kbest")||vm.count("stoch-gen") )) {
			std::cout << "ERROR: you must either train,get kbest,stoch generate data or force-decode\n";
			exit (EXIT_FAILURE);
		}

		if(vm.count("train")) {

			//some basic error checks to parameters
			if(params.learning_rate<=0) {
				std::cout << "ERROR: you cannot have a learning rate <=0\n";
				exit (EXIT_FAILURE);
			}
			if(params.minibatch_size<=0) {
				std::cout << "ERROR: you cannot have a minibatch of size <=0\n";
				exit (EXIT_FAILURE);
			}
			if(params.LSTM_size<=0) {
				std::cout << "ERROR: you cannot have a hiddenstate of size <=0\n";
				exit (EXIT_FAILURE);
			}
			if(params.source_vocab_size<=0) {
				if(params.source_vocab_size!=-1) {
					std::cout << "ERROR: you cannot have a source_vocab_size <=0\n";
					exit (EXIT_FAILURE);
				}
			}
			if(params.target_vocab_size<=0) {
				if(params.target_vocab_size!=-1) {
					std::cout << "ERROR: you cannot have a target_vocab_size <=0\n";
					exit (EXIT_FAILURE);
				}
			}
			if(params.norm_clip<=0) {
				std::cout << "ERROR: you cannot have your norm clip <=0\n";
				exit (EXIT_FAILURE);
			}

			if(params.num_epochs<=0) {
				std::cout << "ERROR: you cannot have num_epochs <=0\n";
				exit (EXIT_FAILURE);
			}

			if(vm.count("truncated-softmax")) {
				params.shortlist_size = std::stoi(trunc_info[0]);
				params.sampled_size = std::stoi(trunc_info[1]);
				params.truncated_softmax = true;
			}


			//now create the necessary files
			if(vm.count("sequence")) {
				
				if(train_files.size()!=2) {
					std::cout << "ERROR: two arguements to be supplied to the train flag"\
					" 1. train data file name, 2. neural network output name\n";
					exit (EXIT_FAILURE);
				}

				params.LM = true;
				params.target_file_name = train_files[0];
				params.output_weight_file = train_files[1];

				input_file_prep input_helper;

				//this outputs the train.txt file along with the mappings and first line
				input_helper.prep_files_train_LM(params.minibatch_size,params.longest_sent,
					params.target_file_name,
					"tmp/train.txt",params.target_vocab_size,
					params.shuffle,params.output_weight_file,params.LSTM_size);
			}
			else {
				//then sequence to sequence model
				if(train_files.size()!=3) {
					std::cout << train_files.size() <<"\n";
					std::cout << "ERROR: three arguements to be supplied to the train flag for the sequence to sequence model\n"\
					" 1. source train data file name\n 2. target train data file name \n3. neural network output name\n";
					exit (EXIT_FAILURE);
				}

				params.LM = false;
				params.source_file_name = train_files[0];
				params.target_file_name = train_files[1];
				params.output_weight_file = train_files[2];

				input_file_prep input_helper;

				input_helper.prep_files_train_nonLM(params.minibatch_size,params.longest_sent,
					params.source_file_name,params.target_file_name,
					"tmp/train.txt",params.source_vocab_size,params.target_vocab_size,
					params.shuffle,params.output_weight_file,params.LSTM_size);

			}

			if(vm.count("parameter-range")) {

				if(lower_upper_range.size()!=2) {
					std::cout << "ERROR: you must have two inputs to parameter-range\n1.lower bound\n2. upper bound\n";
				}

				precision temp_lower = lower_upper_range[0];
				precision temp_upper = lower_upper_range[1];
				if(temp_lower >= temp_upper) {
					std::cout << "ERROR: the lower parameter range cannot be greater than the upper range\n"; 
					exit (EXIT_FAILURE);
				}
				params.lower_range = temp_lower;
				params.upper_range = temp_upper;
			}
				
			if(vm.count("fixed-halve-lr")) {
				params.google_learning_rate = true;
				if(params.epoch_to_start_halving<=0) {
					std::cout << "ERROR: cannot halve learning rate until 1st epoch \n";
					exit (EXIT_FAILURE);
				}
			}

			if(vm.count("adaptive-halve-lr")) {
				params.learning_rate_schedule = true;
				if(vm.count("sequence")) {
					if(adaptive_learning_rate.size()!=1) {
						std::cout << "ERROR: adaptive-halve-lr takes one arguement\n1.dev file name\n";
						exit (EXIT_FAILURE);
					}
					params.dev_target_file_name = adaptive_learning_rate[0];

					input_file_prep input_helper;

					input_helper.integerize_file_LM(params.output_weight_file,params.dev_target_file_name,"tmp/validation.txt",
						params.longest_sent,params.minibatch_size,true,params.LSTM_size,params.target_vocab_size,false,params.source_vocab_size); 

				}
				else {
					if(adaptive_learning_rate.size()!=2) {
						std::cout << "ERROR: adaptive-halve-lr takes two arguements\n1.source dev file name\n2.target dev file name\n";
						exit (EXIT_FAILURE);
					}
					params.dev_source_file_name = adaptive_learning_rate[0];
					params.dev_target_file_name = adaptive_learning_rate[1];

					input_file_prep input_helper;

					input_helper.integerize_file_nonLM(params.output_weight_file,params.dev_source_file_name,
						params.dev_target_file_name,"tmp/validation.txt",
						params.longest_sent,params.minibatch_size,params.LSTM_size,params.source_vocab_size,params.target_vocab_size);
				}
			}

			params.train= true;
			params.decode=false;
			params.test = false;
			params.stochastic_generation = false;
			return;
		}

		if(vm.count("kbest")) {
			if (kbest_files.size()!=4) {
				std::cout << "ERROR: 4 arguements must be entered for kbest, 1. number of best paths"\
				" 2 input file name "
				" 3. neural network file name (this is the output file you get after training the neural network)"\
				" 4. output file name\n";
				exit (EXIT_FAILURE);
			}

			params.num_hypotheses =std::stoi(kbest_files[0]);
			params.decode_tmp_file = kbest_files[1];
			params.input_weight_file = kbest_files[2];
			params.decoder_final_file = kbest_files[3];

			input_file_prep input_helper;

			input_helper.integerize_file_LM(params.input_weight_file,params.decode_tmp_file,"tmp/decoder_input.txt",
				params.longest_sent,1,false,params.LSTM_size,params.target_vocab_size,true,params.source_vocab_size);

			if(params.beam_size<=0) {
				std::cout << "ERROR: beam size cannot be <=0\n";
				exit (EXIT_FAILURE);
			}
			if(params.penalty<0) {
				std::cout << "ERROR: penalty cannot be less than zero\n";
				exit (EXIT_FAILURE);
			}
			params.train = false;
			params.decode = true;
			params.test = false;
			params.stochastic_generation = false;
			params.LM = false;
			return;
		}

		if(vm.count("force-decode")) {
			if(vm.count("sequence")) {
				if(test_files.size()!=3) {
					std::cout << "ERROR: force-decode takes three arguements 1.input file name (input sentences)"\
					"2. neural network file name 3.output file name \n";
					exit (EXIT_FAILURE);
				}

				params.target_file_name = test_files[0];
				params.input_weight_file = test_files[1];
				params.output_force_decode = test_files[2];
				params.LM = true;

				input_file_prep input_helper;

				input_helper.integerize_file_LM(params.input_weight_file,params.target_file_name,"tmp/validation.txt",
					params.longest_sent,1,false,params.LSTM_size,params.target_vocab_size,false,params.source_vocab_size);

			}
			else {
				if(test_files.size()!=4) {
					std::cout << "ERROR: force-decode takes four arguements: 1. source input file"\
					" 2. target input file  3. neural network file name 4. output file name\n";
					exit (EXIT_FAILURE);
				}

				params.LM = false;
				params.source_file_name = test_files[0];
				params.target_file_name = test_files[1];
				params.input_weight_file = test_files[2];
				params.output_force_decode = test_files[3];

				input_file_prep input_helper;

				input_helper.integerize_file_nonLM(params.input_weight_file,params.source_file_name,
					params.target_file_name,"tmp/validation.txt",params.longest_sent,1,params.LSTM_size,
					params.source_vocab_size,params.target_vocab_size);
			}
			params.train= false;
			params.decode=false;
			params.test = true;
			params.minibatch_size=1;
			params.stochastic_generation = false;
			return;
		}

		if(vm.count("stoch-gen")) {
			if(!vm.count("sequence")) {
				std::cout << "ERROR: you can only do stoch-gen on the sequence model\n";
				exit (EXIT_FAILURE);
			}

			if(stoicgen_files.size()!=2) {
				std::cout << "ERROR: stoch-gen takes two inputs"\
				" 1. neural network file name 2. output file name\n";
				exit (EXIT_FAILURE);
			}

			params.input_weight_file = stoicgen_files[0];
			params.sg_output_file = stoicgen_files[1];

			std::ifstream weights_file;
			std::vector<std::string> info;
			std::string str;
			std::string word;
			weights_file.open(params.input_weight_file.c_str());
			weights_file.seekg(0, std::ios::beg);
			std::getline(weights_file, str); //info from first sentence
			std::istringstream iss(str, std::istringstream::in);
			while(iss >> word) {
				info.push_back(word);
			}
			weights_file.close();

			params.LSTM_size = std::stoi(info[1]);
			params.target_vocab_size = std::stoi(info[2]);


			params.LM = true;
			params.train= false;
			params.decode = false;
			params.test = false;
			params.minibatch_size = 1;
			params.stochastic_generation = true;
			return;
		}
	}
	catch(po::error& e) { 
    	std::cerr << "ERROR: " << e.what() << std::endl << std::endl; 
    	//std::cerr << desc << std::endl;
    	exit (EXIT_FAILURE);
    }
}


int main(int argc, char **argv) {

	//Timing stuff
	std::chrono::time_point<std::chrono::system_clock> start_total,
	end_total, begin_minibatch,end_minibatch,begin_decoding,end_decoding;
	std::chrono::duration<double> elapsed_seconds;

    start_total = std::chrono::system_clock::now();

    //Initializing the model
	global_params params; //Declare all of the global parameters
	//file_helper file_info(params.train_file_name,params.minibatch_size,params.train_num_lines_in_file); //Initialize the file information

	//get the command line arguements
	command_line_parse(params,argc,argv);

	neuralMT_model<precision> model; //This is the model
	params.printIntroMessage();
	BZ_CUDA::lower = params.lower_range;
	BZ_CUDA::upper = params.upper_range;

	if(params.google_learning_rate && params.learning_rate_schedule) {
		std::cout << "ERROR: do not select both the fixed learning rate schedule and the perplexity based scheduler";
		std::cout << "I Guarantee this is not what you intended to do\n";
		exit (EXIT_FAILURE);
	}

	if(!params.decode) {
		model.initModel(params.LSTM_size,params.minibatch_size,params.source_vocab_size,params.target_vocab_size,
			params.longest_sent,params.debug,params.learning_rate,params.clip_gradient,params.norm_clip,
			params.input_weight_file,params.output_weight_file,params.softmax_scaled,params.train_perplexity,params.truncated_softmax,
			params.shortlist_size,params.sampled_size,params.LM);
	}

	std::ofstream HPC_output;
	if(params.HPC_output) {
		HPC_output.open("HPC_OUTPUT.txt");
	}

	////////////////////////////////////Train the model//////////////////////////////////////
	if(params.train) {
		//info for averaging the speed
		int curr_batch_num_SPEED = 0;
		const int thres_batch_num_SPEED = 5;//set this to whatever
		int total_words_batch_SPEED = 0;
		double total_batch_time_SPEED = 0;

		//File info for the training file
		file_helper file_info(params.train_file_name,params.minibatch_size,params.train_num_lines_in_file,params.longest_sent,
			params.source_vocab_size,params.target_vocab_size,params.train_total_words,params.truncated_softmax,
			params.shortlist_size,params.sampled_size); //Initialize the file information
		//model.initFileInfo(&file_info);
		params.half_way_count = params.train_total_words/2;
		if(params.google_learning_rate) {
			std::cout << "Words at which to start halving the learning rate: " << params.half_way_count << "\n";
			if(params.HPC_output) {
				HPC_output << "Words at which to start halving the learning rate: " << params.half_way_count << "\n";
				HPC_output.flush();
			}
		}
		int current_epoch = 1;
		std::cout << "Starting model training\n";
		std::cout << "Starting epoch 1\n";
		if(params.HPC_output) {
				HPC_output << "Starting model training\n";
				HPC_output << "Starting epoch 1\n";
				HPC_output.flush();
		}

	
		//stuff for learning rate schedule
		int total_words = 0;
		precision temp_learning_rate = params.learning_rate; //This is only for the google learning rate
		bool learning_rate_flag =true;//used for google learning rate for halving at every 0.5 epochs
		double old_perplexity = 0;
		model.train_perplexity = 0; //set the model perplexity to zero
		while(current_epoch <= params.num_epochs) {
			begin_minibatch = std::chrono::system_clock::now();
			bool success = file_info.read_minibatch();
			end_minibatch = std::chrono::system_clock::now();
			elapsed_seconds = end_minibatch-begin_minibatch;
			//std::cout << "File I/O time: " << elapsed_seconds.count()/60.0 << " minutes\n";
			total_batch_time_SPEED+= elapsed_seconds.count();

			begin_minibatch = std::chrono::system_clock::now();

			//hipProfilerStart();

			model.compute_gradients(file_info.minibatch_tokens_source_input,file_info.minibatch_tokens_source_output,
				file_info.minibatch_tokens_target_input,file_info.minibatch_tokens_target_output,
				file_info.h_input_vocab_indicies_source,file_info.h_output_vocab_indicies_source,
				file_info.h_input_vocab_indicies_target,file_info.h_output_vocab_indicies_target,
				file_info.current_source_length,file_info.current_target_length,
				file_info.h_input_vocab_indicies_source_Wgrad,file_info.h_input_vocab_indicies_target_Wgrad,
				file_info.len_source_Wgrad,file_info.len_target_Wgrad,file_info.h_sampled_indices,
				file_info.len_unique_words_trunc_softmax);

			// hipProfilerStop();
			// return 0;

			end_minibatch = std::chrono::system_clock::now();
			elapsed_seconds = end_minibatch-begin_minibatch;

			total_batch_time_SPEED+= elapsed_seconds.count();
			total_words_batch_SPEED+=file_info.words_in_minibatch;

			if(curr_batch_num_SPEED>=thres_batch_num_SPEED) {
				std::cout << "Batched Minibatch time: " << total_batch_time_SPEED/60.0 << " minutes\n";
				std::cout << "Batched Words in minibatch: " << total_words_batch_SPEED << "\n";
				std::cout << "Batched Throughput: " << (total_words_batch_SPEED)/(total_batch_time_SPEED) << " words per second\n";
				std::cout << total_words << " out of " << params.train_total_words << " epoch: " << current_epoch <<  "\n\n";
				if(params.HPC_output) {
					HPC_output << "Batched Minibatch time: " << total_batch_time_SPEED/60.0 << " minutes\n";
					HPC_output << "Batched Words in minibatch: " << total_words_batch_SPEED << "\n";
					HPC_output << "Batched Throughput: " << (total_words_batch_SPEED)/(total_batch_time_SPEED) << " words per second\n";
					HPC_output << total_words << " out of " << params.train_total_words << " epoch: " << current_epoch <<  "\n\n";
					HPC_output.flush();
				}
				total_words_batch_SPEED = 0;
				total_batch_time_SPEED = 0;
				curr_batch_num_SPEED = 0;

			}
			curr_batch_num_SPEED++;
			total_words += file_info.words_in_minibatch;

			//stuff for google learning rate
			if(params.google_learning_rate && current_epoch>=params.epoch_to_start_halving && total_words>=params.half_way_count &&
				learning_rate_flag) {
					temp_learning_rate = temp_learning_rate/2;
					std::cout << "HALVING LEARNING RATE: " << temp_learning_rate << "\n";
					model.update_learning_rate(temp_learning_rate);
					learning_rate_flag = false;
					if(params.HPC_output) {
						HPC_output << "HALVING LEARNING RATE: " << temp_learning_rate << "\n";
						HPC_output.flush();
					}
			}

			//stuff for perplexity based learning schedule
			if(params.learning_rate_schedule && total_words>=params.half_way_count &&learning_rate_flag) {
				learning_rate_flag = false;
				double new_perplexity = model.get_perplexity(params.test_file_name,params.minibatch_size,params.test_num_lines_in_file,params.longest_sent,
					params.source_vocab_size,params.target_vocab_size,HPC_output,false,params.test_total_words,params.HPC_output,false,"");
				std::cout << "Old dev set Perplexity: " << old_perplexity << "\n";
				std::cout << "New dev set Perplexity: " << new_perplexity << "\n";
				if(params.HPC_output) {
					HPC_output << "Old dev set Perplexity: " << old_perplexity << "\n";
					HPC_output << "New dev set Perplexity: " << new_perplexity << "\n";
					HPC_output.flush();
				}
				if ( (new_perplexity + params.margin >= old_perplexity) && current_epoch!=1) {
					temp_learning_rate = temp_learning_rate*params.decrease_factor;
					model.update_learning_rate(temp_learning_rate);
					std::cout << "Halving learning rate!\n\n";
					if(params.HPC_output) {
						HPC_output << "Halving learning rate!\n\n";
						HPC_output.flush();
					}
				}
				old_perplexity = new_perplexity;
			}

			if(!success) {
				current_epoch+=1;
				//stuff for google learning rate schedule
				if(params.google_learning_rate && current_epoch>=params.epoch_to_start_halving) {
					temp_learning_rate = temp_learning_rate/2;
					std::cout << "HALVING LEARNING RATE: " << temp_learning_rate << "\n";
					model.update_learning_rate(temp_learning_rate);
					learning_rate_flag = true;
					if(params.HPC_output) {
						HPC_output << "HALVING LEARNING RATE: " << temp_learning_rate << "\n";
						HPC_output.flush();
					}
				}

				double new_perplexity;
				if(params.google_learning_rate || params.learning_rate_schedule) {
					new_perplexity = model.get_perplexity(params.test_file_name,params.minibatch_size,params.test_num_lines_in_file,params.longest_sent,
						params.source_vocab_size,params.target_vocab_size,HPC_output,false,params.test_total_words,params.HPC_output,false,"");
				}
				//stuff for perplexity based learning schedule
				if(params.learning_rate_schedule) {
					std::cout << "Old dev set Perplexity: " << old_perplexity << "\n";
					std::cout << "New dev set Perplexity: " << new_perplexity << "\n";
					if(params.HPC_output) {
						HPC_output << "Old dev set Perplexity: " << old_perplexity << "\n";
						HPC_output << "New dev set Perplexity: " << new_perplexity << "\n";
						HPC_output.flush();
					}
					if ( (new_perplexity + params.margin >= old_perplexity) && current_epoch!=1) {
						std::cout << "Halving learning rate!\n\n";
						if(params.HPC_output) {
							HPC_output << "Halving learning rate!\n\n";
							HPC_output.flush();
						}
						temp_learning_rate = temp_learning_rate*params.decrease_factor;
						model.update_learning_rate(temp_learning_rate);
					}
					learning_rate_flag = true;
					old_perplexity = new_perplexity;
				}

				if(params.train_perplexity) {
					std::cout << "PData on train set:"  << model.train_perplexity << "\n";
					std::cout << "Total target words: " << file_info.total_target_words << "\n";
					std::cout << "Training set perplexity: " << std::pow(2,-1*model.train_perplexity/file_info.total_target_words) << "\n";
					if(params.HPC_output) {
						HPC_output << "Training set perplexity: " << std::pow(2,-1*model.train_perplexity/file_info.total_target_words) << "\n";
						HPC_output.flush();
					}
					model.train_perplexity = 0;
				}

				total_words=0;
				if(current_epoch <= params.num_epochs) {
					std::cout << "-----------------------------------"  << std::endl;
					std::cout << "Starting epoch " << current_epoch << std::endl;
					std::cout << "-----------------------------------"  << std::endl;
					if(params.HPC_output) {
						HPC_output << "Starting epoch " << current_epoch << std::endl;
						HPC_output.flush();
					}
				}
			}
			hipDeviceSynchronize();
		}	
		//Now that training is done, dump the weights
		hipDeviceSynchronize();
		model.dump_weights();
	}


	/////////////////////////////////Get perplexity on test set////////////////////////////////
	if(params.test) {
		model.get_perplexity(params.test_file_name,params.minibatch_size,params.test_num_lines_in_file,params.longest_sent,
			params.source_vocab_size,params.target_vocab_size,HPC_output,true,params.test_total_words,params.HPC_output,true,params.output_force_decode);
	}

	if(params.LM && params.stochastic_generation) {
		model.stoicastic_generation(params.sg_length,params.sg_output_file_temp);
		input_file_prep input_helper;
		input_helper.unint_file(params.input_weight_file,"tmp/sg.txt",params.sg_output_file,true,false);
	}


	///////////////////////////////////////////decode the model////////////////////////////////////////////
	if(params.decode) {
		std::cout << "-----------------Starting Decoding----------------\n";
		begin_decoding = std::chrono::system_clock::now();
		model.beam_decoder(params.beam_size,params.decode_file_name,
			params.input_weight_file,params.decode_num_lines_in_file,params.source_vocab_size,
			params.target_vocab_size,params.longest_sent,params.LSTM_size,params.penalty,
			params.decoder_output_file,params.min_decoding_ratio,params.max_decoding_ratio,params.softmax_scaled,
			params.num_hypotheses,params.print_score);
		end_decoding = std::chrono::system_clock::now();
		std::chrono::duration<double> elapsed_seconds = end_decoding-begin_decoding;
		std::cout << "Decoding time: " << elapsed_seconds.count()/60.0 << " minutes\n";

		//now unintegerize the file
		input_file_prep input_helper;
		input_helper.unint_file(params.input_weight_file,"tmp/decoder_output.txt",params.decoder_final_file,false,true);
	}

	//Compute the final runtime
	end_total = std::chrono::system_clock::now();
	elapsed_seconds = end_total-start_total;
    std::cout << "\n\n\n";
    std::cout << "Total Program Runtime: " << elapsed_seconds.count()/60.0 << " minutes" << std::endl;
}
