#include "hip/hip_runtime.h"
//STL includes
#include <iostream>
#include <vector>
#include <time.h>
#include <cmath>
#include <chrono>
#include <iomanip>

//Eigen includes
#include <Eigen/Dense>
#include <Eigen/Sparse>

//Boost
#include "boost/program_options.hpp" 
#include <boost/filesystem/operations.hpp>
#include <boost/filesystem/path.hpp>

//My own includes
#include "global_params.h"
#include "input_file_prep.h"
#include "BZ_CUDA_UTIL.h"
#include "base_layer.h"
#include "gpu_info_struct.h"
#include "custom_kernels.h"
#include "model.h"
#include "fileHelper.h"
#include "Eigen_Util.h"
#include "model.hpp"
#include "base_layer.hpp"
#include "LSTM.hpp"
#include "softmax.hpp"
#include "Input_To_Hidden_Layer.hpp"
#include "Hidden_To_Hidden_Layer.hpp"
#include "LSTM_HH.hpp"

//parse the command line from the user
void command_line_parse(global_params &params,int argc, char **argv) {
		
	//files for keeping the user input
	//if not s, 1st source, 2nd target, 3rd output weights name
	//if s, 1st target, 2nd output weights name
	std::vector<std::string> train_files;

	//files for force decoding
	//if not s, 1. source input file 2. target input file  3. neural network file name 4. output file name
	//if s, 1. target input file  2. neural network file name 3. output file name
	std::vector<std::string> test_files;

	//stuff for adaptive learning rate schedule
	//if not seq , 1st is source dev, 2nd is target dev
	//if seq 1st is target dev
	std::vector<std::string> adaptive_learning_rate;

	//lower and upper range for parameter initialization
	std::vector<precision> lower_upper_range;

	//for the kbest flag, 4 arguements must be entered for kbest, 1. number of best paths 2 input file name 
	//3. neural network file name (this is the output file you get after training the neural network)4. output file name
	std::vector<std::string> kbest_files;

	//for stoic gen, 1st neural network file, 2nd is output file name
	std::vector<std::string> stoicgen_files;

	//truncated softmax
	std::vector<std::string> trunc_info;

	//for decoding ratios 
	std::vector<precision> decoding_ratio;

	//for continuing to train
	std::vector<std::string> cont_train;

	//basic format setup
	namespace po = boost::program_options; 
	po::options_description desc("Options");
	desc.add_options() 
  		("help,h", "Run to get help on how to use the program")
  		("train,t",po::value<std::vector<std::string> > (&train_files)->multitoken(),"Train a model with input data file(s) and a name for the neural network output file"\
  			". \nFORMAT (if sequence to sequence): <source file name> <target file name> <neural network output name> "\
  			" \nFORMAT (if sequence): <target file name> <neural network output name>")
  		("cont-train,C",po::value<std::vector<std::string>> (&cont_train)->multitoken(),"Resume training of a model (THIS WILL OVERWRITE THE MODEL FILE)\n"\
  			"FORMAT: (if sequence to sequence): <source file name> <target file name> <neural network file name>\n"\
  			"FORMAT: (if seq): <target file name> <neural network file name>")
  		("force-decode,f",po::value<std::vector<std::string> > (&test_files)->multitoken(), "Get per line probability of dataset plus the perplexity\n"\
  			"FORMAT: (if sequence to sequence): <source file name> <target file name> <trained neural network file name> <output file name>\n"\
  			"FORMAT: (if sequence): <target file name> <trained neural network file name> <output file name>")
  		("stoch-gen,g", po::value<std::vector<std::string> > (&stoicgen_files)->multitoken(),"Do random generation for a sequence model, such as a language model\n"\
  			"FORMAT: <neural network file name> <output file name>")
  		("stoch-gen-len",po::value<int>(&params.sg_length) ,"How many sentences to let stoch-gen run for\n"\
  			"FORMAT: <num sentences>\n"
  			"DEFAULT: 100")
  		("temperature",po::value<double>(&params.temperature) ,"What should the temperature be for the stoch generation"\
  			"FORMAT: <temperature>  where temperature is typically between [0,1]. A lower temperature makes the model output less and less from what it memorized from training\n"\
  			"DEFAULT: 1")
  		("sequence,s", "Train model that learns a sequence,such as language modeling. Default model is sequence to sequence model")
  		("learning-rate,l",po::value<precision>(&params.learning_rate),"Set the learning rate\n DEFAULT: 0.7")
  		("longest-sent,L",po::value<int>(&params.longest_sent),"Set the maximum sentence length for training.\n DEFAULT: 100")
  		("hiddenstate-size,H",po::value<int>(&params.LSTM_size),"Set hiddenstate size \n DEFAULT: 1000")
  		("truncated-softmax,T",po::value<std::vector<std::string>> (&trunc_info)->multitoken(),"Use truncated softmax\n DEFAULT: not being used\n"\
  			"FORMAT: <shortlist size> <sampled size>")
  		("source-vocab,v",po::value<int>(&params.source_vocab_size),"Set source vocab size\n DEFAULT: number of unique words in source training corpus")
  		("target-vocab,V",po::value<int>(&params.target_vocab_size),"Set target vocab size\n DEFAULT: number of unique words in target training corpus")
  		("shuffle",po::value<bool>(&params.shuffle),"true if you want to shuffle the train data\n DEFAULT: true")
  		("parameter-range,P",po::value<std::vector<precision> > (&lower_upper_range)->multitoken(),"parameter initialization range\n"\
  			"FORMAT: <Lower range value> <Upper range value>\n DEFAULT: -0.08 0.08")
  		("number-epochs,n",po::value<int>(&params.num_epochs),"Set number of epochs\n DEFAULT: 10")
  		("clip-gradients,c",po::value<precision>(&params.norm_clip),"Set gradient clipping threshold\n DEFAULT: 5")
  		("adaptive-halve-lr,a",po::value<std::vector<std::string>> (&adaptive_learning_rate)->multitoken(),"change the learning rate"\
  			" when the perplexity on your specified dev set decreases from the previous half epoch by some constant, so "\
  			" new_learning_rate = constant*old_learning rate, by default the constant is 0.5, but can be set using adaptive-decrease-factor\n"
  			"FORMAT: (if sequence to sequence): <source dev file name> <target dev file name>\n"\
  			"FORMAT: (if sequence): <target dev file name>")
  		("adaptive-decrease-factor,A",po::value<precision>(&params.decrease_factor),"To be used with adaptive-halve-lr"\
  			" it\n DEFAULT: 0.5")
  		("fixed-halve-lr",po::value<int> (&params.epoch_to_start_halving),"Halve the learning rate"\
  			" after a certain epoch, every half epoch afterwards by a specific amount")
  		("minibatch-size,m",po::value<int>(&params.minibatch_size),"Set minibatch size\n DEFAULT: 128")
  		("screen-print-rate",po::value<int>(&params.screen_print_rate),"Set after how many minibatched you want to print training info to the screen\n DEFAULT: 5")
  		("HPC-output",po::value<std::string>(&params.HPC_output_file_name),"Use if you want to have the terminal output also be put to a" \
  			"file \n FORMAT: <file name>")
  		("best-model,B",po::value<std::string>(&params.best_model_file_name),"During train have the best model be written to a file\nFORMAT: <output file name>")
  		("kbest,k",po::value<std::vector<std::string> > (&kbest_files)->multitoken(),"Get k best paths in sequence to sequence model\n"\
  			"FORMAT: <how many paths> <source file name> <neural network file name> <output file name>") 
  		("beam-size,b",po::value<int>(&params.beam_size),"Set beam size for kbest paths\n DEFAULT: 12")
  		("penalty,p",po::value<precision>(&params.penalty),"Set penalty for kbest decoding. The value entered"\
  			" will be added to the log probability score per target word decoded. This can make the model favor longer sentences for decoding\n DEFAULT: 0")
  		("print-score",po::value<bool>(&params.print_score),"Set if you want to print out the unnormalized log prob for each path "\
  			"FORMAT: <bool> \nthe bool is 1 if you want to print the score or 0 otherwise.\n DEFAULT: false")
  		("dec-ratio",po::value<std::vector<precision>>(&decoding_ratio)->multitoken(),"Set the min and max decoding length rations\n"\
  			"This means that a target decoded sentence must be at least min_dec_ratio*len(source sentence)"\
  			" and not longer than max_dec_ratio*len(source sentence)\nFORMAT: <min ration> <max ratio>\n"\
  			"DEFAULT: 0.5, 1.5")
  		("Dump-LSTM",po::value<std::string>(&params.LSTM_dump_file),"Print the output at each timestep from the LSTM\nFORMAT: <output file name>\n"\
  			"The file lines that are output are the following: 1.input word, embedding   2.Forget gate   3.input gate"\
  			"   4.c_t   5.output gate    6.h_t     7.probabilities");

    po::variables_map vm; 

	try {
		po::store(po::parse_command_line(argc, argv, desc), vm);
		po::notify(vm);
		
		//see if the user specified the help flag
		if ( vm.count("help") ) {

			std::cout << "\n------------------------------\n";
			std::cout << "This is Barret Zoph's GPU RNN library\n"
            << "The flags for the command line interface are below\n" 
            << "" << "\n";

			std::cout << desc << "\n";
			exit (EXIT_FAILURE);
		}

		//error checks to be sure only once of these options is set
		if (vm.count("train") && vm.count("kbest")) {
			std::cout << "ERROR: you cannot train and get kbest at the same time\n";
			exit (EXIT_FAILURE);
		}
		if (vm.count("train") && vm.count("force-decode")) {
			std::cout << "ERROR: you cannot train and force-decode at the same time\n";
			exit (EXIT_FAILURE);
		}
		if (vm.count("force-decode") && vm.count("kbest")) {
			std::cout << "ERROR: you cannot force-decode and get kbest at the same time\n";
			exit (EXIT_FAILURE);
		}
		if (!(vm.count("train") || vm.count("force-decode") || vm.count("kbest")||vm.count("stoch-gen") || vm.count("cont-train") )) {
			std::cout << "ERROR: you must either train,continue training,get kbest,stoch generate data or force-decode\n";
			exit (EXIT_FAILURE);
		}

		params.longest_sent+=4; //because it is really 4 less

		if(vm.count("train") || vm.count("cont-train")) {

			//some basic error checks to parameters
			if(params.learning_rate<=0) {
				std::cout << "ERROR: you cannot have a learning rate <=0\n";
				exit (EXIT_FAILURE);
			}
			if(params.minibatch_size<=0) {
				std::cout << "ERROR: you cannot have a minibatch of size <=0\n";
				exit (EXIT_FAILURE);
			}
			if(params.LSTM_size<=0) {
				std::cout << "ERROR: you cannot have a hiddenstate of size <=0\n";
				exit (EXIT_FAILURE);
			}
			if(params.source_vocab_size<=0) {
				if(params.source_vocab_size!=-1) {
					std::cout << "ERROR: you cannot have a source_vocab_size <=0\n";
					exit (EXIT_FAILURE);
				}
			}
			if(params.target_vocab_size<=0) {
				if(params.target_vocab_size!=-1) {
					std::cout << "ERROR: you cannot have a target_vocab_size <=0\n";
					exit (EXIT_FAILURE);
				}
			}
			if(params.norm_clip<=0) {
				std::cout << "ERROR: you cannot have your norm clip <=0\n";
				exit (EXIT_FAILURE);
			}

			if(params.num_epochs<=0) {
				std::cout << "ERROR: you cannot have num_epochs <=0\n";
				exit (EXIT_FAILURE);
			}

			if(vm.count("HPC-output")) {
				params.HPC_output = true;
			}


			boost::filesystem::path unique_path = boost::filesystem::unique_path();
			std::cout << "Temp directory being created named: " << unique_path.string() << "\n";
			boost::filesystem::create_directories(unique_path);
			params.unique_dir = unique_path.string();

			params.train_file_name = params.unique_dir+"/train.txt";


			if(vm.count("cont-train")) {

				//sequence model
				if(vm.count("sequence")) {
					if(cont_train.size()!=2) {
						std::cout << cont_train.size() << "\n";
						std::cout << "ERROR: two arguements to be supplied to the continue train flag\n"\
						" 1. train data file name, 2. neural network file name\n";
						boost::filesystem::path temp_path(params.unique_dir);
						boost::filesystem::remove_all(temp_path);
						exit (EXIT_FAILURE);
					}


					params.target_file_name = cont_train[0];
					params.input_weight_file = cont_train[1];
					params.output_weight_file = cont_train[1];
					params.LM = true;
					params.load_model_train = true;
					params.load_model_name = params.input_weight_file;

					input_file_prep input_helper;

					input_helper.integerize_file_LM(params.input_weight_file,params.target_file_name,params.train_file_name,
						params.longest_sent,params.minibatch_size,true,params.LSTM_size,params.target_vocab_size);

				}
				else {
					if(cont_train.size()!=3) {
						std::cout << "ERROR: three arguements to be supplied to the continue train flag\n"\
						" 1. source train data file name  2. target train data file name  3. neural network file name  \n";
						boost::filesystem::path temp_path(params.unique_dir);
						boost::filesystem::remove_all(temp_path);
						exit (EXIT_FAILURE);
					}

					params.LM = false;
					params.source_file_name = cont_train[0];
					params.target_file_name = cont_train[1];
					params.input_weight_file = cont_train[2];
					params.output_weight_file = cont_train[2];
					params.load_model_train = true;
					params.load_model_name = params.input_weight_file;

					if(params.source_file_name == params.target_file_name) {
						std::cout << "ERROR: do not use the same file for source and target data\n";
						boost::filesystem::path temp_path(params.unique_dir);
						boost::filesystem::remove_all(temp_path);
						exit (EXIT_FAILURE);
					}

					input_file_prep input_helper;

					input_helper.integerize_file_nonLM(params.input_weight_file,params.source_file_name,
						params.target_file_name,params.train_file_name,params.longest_sent,params.minibatch_size,params.LSTM_size,
						params.source_vocab_size,params.target_vocab_size);
				}
			}
			else {

				//now create the necessary files
				if(vm.count("sequence")) {
					
					if(train_files.size()!=2) {
						std::cout << "ERROR: two arguements to be supplied to the train flag"\
						" 1. train data file name, 2. neural network output name\n";
						boost::filesystem::path temp_path(params.unique_dir);
						boost::filesystem::remove_all(temp_path);
						exit (EXIT_FAILURE);
					}

					params.LM = true;
					params.target_file_name = train_files[0];
					params.output_weight_file = train_files[1];

					input_file_prep input_helper;

					//this outputs the train.txt file along with the mappings and first line
					bool success = input_helper.prep_files_train_LM(params.minibatch_size,params.longest_sent,
						params.target_file_name,
						params.train_file_name,params.target_vocab_size,
						params.shuffle,params.output_weight_file,params.LSTM_size);

					//clean up if error
					if(!success) {
						boost::filesystem::path temp_path(params.unique_dir);
						boost::filesystem::remove_all(temp_path);
						exit (EXIT_FAILURE);
					}
				}
				else {
					//then sequence to sequence model
					if(train_files.size()!=3) {
						std::cout << train_files.size() <<"\n";
						std::cout << "ERROR: three arguements to be supplied to the train flag for the sequence to sequence model\n"\
						" 1. source train data file name\n 2. target train data file name \n3. neural network output name\n";
						boost::filesystem::path temp_path(params.unique_dir);
						boost::filesystem::remove_all(temp_path);
						exit (EXIT_FAILURE);
					}

					params.LM = false;
					params.source_file_name = train_files[0];
					params.target_file_name = train_files[1];
					params.output_weight_file = train_files[2];

					if(params.source_file_name == params.target_file_name) {
						std::cout << "ERROR: do not use the same file for source and target data\n";
						boost::filesystem::path temp_path(params.unique_dir);
						boost::filesystem::remove_all(temp_path);
						exit (EXIT_FAILURE);
					}

					input_file_prep input_helper;

					bool success = input_helper.prep_files_train_nonLM(params.minibatch_size,params.longest_sent,
						params.source_file_name,params.target_file_name,
						params.train_file_name,params.source_vocab_size,params.target_vocab_size,
						params.shuffle,params.output_weight_file,params.LSTM_size);
					//clean up if error
					if(!success) {
						boost::filesystem::path temp_path(params.unique_dir);
						boost::filesystem::remove_all(temp_path);
						exit (EXIT_FAILURE);
					}
				}
			}

			if(vm.count("parameter-range")) {

				if(lower_upper_range.size()!=2) {
					std::cout << "ERROR: you must have two inputs to parameter-range\n1.lower bound\n2. upper bound\n";
					boost::filesystem::path temp_path(params.unique_dir);
					boost::filesystem::remove_all(temp_path);
					exit (EXIT_FAILURE);
				}

				precision temp_lower = lower_upper_range[0];
				precision temp_upper = lower_upper_range[1];
				if(temp_lower >= temp_upper) {
					std::cout << "ERROR: the lower parameter range cannot be greater than the upper range\n";
					boost::filesystem::path temp_path(params.unique_dir);
					boost::filesystem::remove_all(temp_path);
					exit (EXIT_FAILURE);
				}
				params.lower_range = temp_lower;
				params.upper_range = temp_upper;
			}
				
			if(vm.count("fixed-halve-lr")) {
				params.google_learning_rate = true;
				if(params.epoch_to_start_halving<=0) {
					std::cout << "ERROR: cannot halve learning rate until 1st epoch \n";
					boost::filesystem::path temp_path(params.unique_dir);
					boost::filesystem::remove_all(temp_path);
					exit (EXIT_FAILURE);
				}
			}

			if(vm.count("adaptive-halve-lr")) {
				params.learning_rate_schedule = true;
				if(vm.count("sequence")) {
					if(adaptive_learning_rate.size()!=1) {
						std::cout << "ERROR: adaptive-halve-lr takes one arguement\n1.dev file name\n";
						boost::filesystem::path temp_path(params.unique_dir);
						boost::filesystem::remove_all(temp_path);
						exit (EXIT_FAILURE);
					}
					params.dev_target_file_name = adaptive_learning_rate[0];

					input_file_prep input_helper;

					input_helper.integerize_file_LM(params.output_weight_file,params.dev_target_file_name,"tmp/validation.txt",
						params.longest_sent,params.minibatch_size,true,params.LSTM_size,params.target_vocab_size); 

				}
				else {
					if(adaptive_learning_rate.size()!=2) {
						std::cout << "ERROR: adaptive-halve-lr takes two arguements\n1.source dev file name\n2.target dev file name\n";
						boost::filesystem::path temp_path(params.unique_dir);
						boost::filesystem::remove_all(temp_path);
						exit (EXIT_FAILURE);
					}
					params.dev_source_file_name = adaptive_learning_rate[0];
					params.dev_target_file_name = adaptive_learning_rate[1];

					if(params.dev_source_file_name == params.dev_target_file_name) {
						std::cout << "ERROR: do not use the same file for source and target data\n";
						boost::filesystem::path temp_path(params.unique_dir);
						boost::filesystem::remove_all(temp_path);
						exit (EXIT_FAILURE);
					}

					input_file_prep input_helper;

					input_helper.integerize_file_nonLM(params.output_weight_file,params.dev_source_file_name,
						params.dev_target_file_name,"tmp/validation.txt",
						params.longest_sent,params.minibatch_size,params.LSTM_size,params.source_vocab_size,params.target_vocab_size);
				}

				if(vm.count("best-model")) {
					params.best_model = true;
				}
			}

			if(vm.count("truncated-softmax")) {
				params.shortlist_size = std::stoi(trunc_info[0]);
				params.sampled_size = std::stoi(trunc_info[1]);
				params.truncated_softmax = true;
				if(params.shortlist_size + params.sampled_size > params.target_vocab_size) {
					std::cout << "ERROR: you cannot have shortlist size + sampled size >= target vocab size\n";
					boost::filesystem::path temp_path(params.unique_dir);
					boost::filesystem::remove_all(temp_path);
					exit (EXIT_FAILURE);
				}
			}

			params.train= true;
			params.decode=false;
			params.test = false;
			params.stochastic_generation = false;
			return;
		}

		if(vm.count("kbest")) {
			if (kbest_files.size()!=4) {
				std::cout << "ERROR: 4 arguements must be entered for kbest, 1. number of best paths"\
				" 2 input file name "
				" 3. neural network file name (this is the output file you get after training the neural network)"\
				" 4. output file name\n";
				exit (EXIT_FAILURE);
			}

			boost::filesystem::path unique_path = boost::filesystem::unique_path();
			std::cout << "Temp directory being created named: " << unique_path.string() << "\n";
			boost::filesystem::create_directories(unique_path);
			params.unique_dir = unique_path.string();

			params.decode_file_name = params.unique_dir+"/decoder_input.txt";
			params.decoder_output_file = params.unique_dir+"/decoder_output.txt";

			params.num_hypotheses =std::stoi(kbest_files[0]);
			params.decode_tmp_file = kbest_files[1];
			params.input_weight_file = kbest_files[2];
			params.decoder_final_file = kbest_files[3];

			input_file_prep input_helper;

			// input_helper.integerize_file_LM(params.input_weight_file,params.decode_tmp_file,"tmp/decoder_input.txt",
			// 	params.longest_sent,1,false,params.LSTM_size,params.target_vocab_size,true,params.source_vocab_size);

			input_helper.integerize_file_kbest(params.input_weight_file,params.decode_tmp_file,params.decode_file_name,
				params.longest_sent,params.LSTM_size,params.target_vocab_size,params.source_vocab_size);

			if(params.beam_size<=0) {
				std::cout << "ERROR: beam size cannot be <=0\n";
				boost::filesystem::path temp_path(params.unique_dir);
				boost::filesystem::remove_all(temp_path);
				exit (EXIT_FAILURE);
			}
			if(params.penalty<0) {
				std::cout << "ERROR: penalty cannot be less than zero\n";
				boost::filesystem::path temp_path(params.unique_dir);
				boost::filesystem::remove_all(temp_path);
				exit (EXIT_FAILURE);
			}

			if(vm.count("Dump-LSTM")) {
				params.dump_LSTM=true;
			}

			if(vm.count("dec-ratio")) {
				if(decoding_ratio.size()!=2) {
					std::cout << "Decoding ratio size: " << decoding_ratio.size() << "\n";
					std::cout << decoding_ratio[0] << "\n";
					std::cout << "ERROR: only two inputs for decoding ratio\n";
					boost::filesystem::path temp_path(params.unique_dir);
					boost::filesystem::remove_all(temp_path);
					exit (EXIT_FAILURE);
				}
				params.min_decoding_ratio = decoding_ratio[0];
				params.max_decoding_ratio = decoding_ratio[1];
				if(params.min_decoding_ratio >= params.max_decoding_ratio) {
					std::cout << "ERROR: min decoding ratio must be <= max_decoding_ratio\n";
					boost::filesystem::path temp_path(params.unique_dir);
					boost::filesystem::remove_all(temp_path);
					exit (EXIT_FAILURE);
				}
			}

			params.train = false;
			params.decode = true;
			params.test = false;
			params.stochastic_generation = false;
			params.LM = false;
			return;
		}

		if(vm.count("force-decode")) {

			boost::filesystem::path unique_path = boost::filesystem::unique_path();
			std::cout << "Temp directory being created named: " << unique_path.string() << "\n";
			boost::filesystem::create_directories(unique_path);
			params.unique_dir = unique_path.string();
			params.test_file_name = params.unique_dir + "/validation.txt";

			if(vm.count("sequence")) {
				if(test_files.size()!=3) {
					std::cout << "ERROR: force-decode takes three arguements 1.input file name (input sentences)"\
					"2. neural network file name 3.output file name \n";
					boost::filesystem::path temp_path(params.unique_dir);
					boost::filesystem::remove_all(temp_path);
					exit (EXIT_FAILURE);
				}

				params.target_file_name = test_files[0];
				params.input_weight_file = test_files[1];
				params.output_force_decode = test_files[2];
				params.LM = true;

				input_file_prep input_helper;

				input_helper.integerize_file_LM(params.input_weight_file,params.target_file_name,params.test_file_name,
					params.longest_sent,1,false,params.LSTM_size,params.target_vocab_size);

			}
			else {
				if(test_files.size()!=4) {
					std::cout << "ERROR: force-decode takes four arguements: 1. source input file"\
					" 2. target input file  3. neural network file name 4. output file name\n";
					boost::filesystem::path temp_path(params.unique_dir);
					boost::filesystem::remove_all(temp_path);
					exit (EXIT_FAILURE);
				}

				params.LM = false;
				params.source_file_name = test_files[0];
				params.target_file_name = test_files[1];
				params.input_weight_file = test_files[2];
				params.output_force_decode = test_files[3];

				if(params.source_file_name == params.target_file_name) {
					std::cout << "ERROR: do not use the same file for source and target data\n";
					boost::filesystem::path temp_path(params.unique_dir);
					boost::filesystem::remove_all(temp_path);
					exit (EXIT_FAILURE);
				}

				input_file_prep input_helper;

				input_helper.integerize_file_nonLM(params.input_weight_file,params.source_file_name,
					params.target_file_name,params.test_file_name,params.longest_sent,1,params.LSTM_size,
					params.source_vocab_size,params.target_vocab_size);
			}
			params.train= false;
			params.decode=false;
			params.test = true;
			params.minibatch_size=1;
			params.stochastic_generation = false;
			return;
		}

		if(vm.count("stoch-gen")) {
			if(!vm.count("sequence")) {
				std::cout << "ERROR: you can only do stoch-gen on the sequence model\n";
				exit (EXIT_FAILURE);
			}

			if(stoicgen_files.size()!=2) {
				std::cout << "ERROR: stoch-gen takes two inputs"\
				" 1. neural network file name 2. output file name\n";
				exit (EXIT_FAILURE);
			}

			boost::filesystem::path unique_path = boost::filesystem::unique_path();
			std::cout << "Temp directory being created named: " << unique_path.string() << "\n";
			boost::filesystem::create_directories(unique_path);
			params.unique_dir = unique_path.string();
			params.sg_output_file_temp = params.unique_dir + "/sg.txt";

			params.input_weight_file = stoicgen_files[0];
			params.sg_output_file = stoicgen_files[1];

			std::ifstream weights_file;
			std::vector<std::string> info;
			std::string str;
			std::string word;
			weights_file.open(params.input_weight_file.c_str());
			weights_file.seekg(0, std::ios::beg);
			std::getline(weights_file, str); //info from first sentence
			std::istringstream iss(str, std::istringstream::in);
			while(iss >> word) {
				info.push_back(word);
			}
			weights_file.close();

			params.LSTM_size = std::stoi(info[1]);
			params.target_vocab_size = std::stoi(info[2]);


			params.LM = true;
			params.train= false;
			params.decode = false;
			params.test = false;
			params.minibatch_size = 1;
			params.stochastic_generation = true;
			return;
		}
	}
	catch(po::error& e) { 
    	std::cerr << "ERROR: " << e.what() << std::endl << std::endl; 
    	//std::cerr << desc << std::endl;
    	exit (EXIT_FAILURE);
    }
}



int main(int argc, char **argv) {

	//Timing stuff
	std::chrono::time_point<std::chrono::system_clock> start_total,
	end_total, begin_minibatch,end_minibatch,begin_decoding,end_decoding;
	std::chrono::duration<double> elapsed_seconds;

    start_total = std::chrono::system_clock::now();

    //Initializing the model
	global_params params; //Declare all of the global parameters

    //create tmp directory if it does not exist already
	// if( !(boost::filesystem::exists("tmp/"))) {
	//     std::cout << "Creating tmp directory for program\n";
	//     boost::filesystem::create_directory("tmp/");
	// }



	//file_helper file_info(params.train_file_name,params.minibatch_size,params.train_num_lines_in_file); //Initialize the file information

	//get the command line arguements
	command_line_parse(params,argc,argv);

	neuralMT_model<precision> model; //This is the model
	params.printIntroMessage();
	BZ_CUDA::lower = params.lower_range;
	BZ_CUDA::upper = params.upper_range;

	if(params.google_learning_rate && params.learning_rate_schedule) {
		std::cout << "ERROR: do not select both the fixed learning rate schedule and the perplexity based scheduler";
		std::cout << "I Guarantee this is not what you intended to do\n";
		exit (EXIT_FAILURE);
	}

	if(!params.decode) {
		model.initModel(params.LSTM_size,params.minibatch_size,params.source_vocab_size,params.target_vocab_size,
			params.longest_sent,params.debug,params.learning_rate,params.clip_gradient,params.norm_clip,
			params.input_weight_file,params.output_weight_file,params.softmax_scaled,params.train_perplexity,params.truncated_softmax,
			params.shortlist_size,params.sampled_size,params.LM);
	}

	if(params.load_model_train) {
		std::string temp_swap_weights = model.input_weight_file;
		model.input_weight_file = params.load_model_name;
		model.load_weights();
		model.input_weight_file = temp_swap_weights;
	}

	std::ofstream HPC_output;
	if(params.HPC_output) {
		HPC_output.open("HPC_OUTPUT.txt");
	}

	////////////////////////////////////Train the model//////////////////////////////////////
	if(params.train) {
		//info for averaging the speed
		int curr_batch_num_SPEED = 0;
		const int thres_batch_num_SPEED = params.screen_print_rate;//set this to whatever
		int total_words_batch_SPEED = 0;
		double total_batch_time_SPEED = 0;

		//File info for the training file
		file_helper file_info(params.train_file_name,params.minibatch_size,params.train_num_lines_in_file,params.longest_sent,
			params.source_vocab_size,params.target_vocab_size,params.train_total_words,params.truncated_softmax,
			params.shortlist_size,params.sampled_size); //Initialize the file information
		//model.initFileInfo(&file_info);
		params.half_way_count = params.train_total_words/2;
		if(params.google_learning_rate) {
			std::cout << "Words at which to start halving the learning rate: " << params.half_way_count << "\n";
			if(params.HPC_output) {
				HPC_output << "Words at which to start halving the learning rate: " << params.half_way_count << "\n";
				HPC_output.flush();
			}
		}
		int current_epoch = 1;
		std::cout << "Starting model training\n";
		std::cout << "Starting epoch 1\n";
		if(params.HPC_output) {
				HPC_output << "Starting model training\n";
				HPC_output << "Starting epoch 1\n";
				HPC_output.flush();
		}

	
		//stuff for learning rate schedule
		int total_words = 0;
		precision temp_learning_rate = params.learning_rate; //This is only for the google learning rate
		bool learning_rate_flag =true;//used for google learning rate for halving at every 0.5 epochs
		double old_perplexity = 0;
		model.train_perplexity = 0; //set the model perplexity to zero
		while(current_epoch <= params.num_epochs) {
			begin_minibatch = std::chrono::system_clock::now();
			bool success = file_info.read_minibatch();
			end_minibatch = std::chrono::system_clock::now();
			elapsed_seconds = end_minibatch-begin_minibatch;
			//std::cout << "File I/O time: " << elapsed_seconds.count()/60.0 << " minutes\n";
			total_batch_time_SPEED+= elapsed_seconds.count();

			begin_minibatch = std::chrono::system_clock::now();

			//hipProfilerStart();

			model.compute_gradients(file_info.minibatch_tokens_source_input,file_info.minibatch_tokens_source_output,
				file_info.minibatch_tokens_target_input,file_info.minibatch_tokens_target_output,
				file_info.h_input_vocab_indicies_source,file_info.h_output_vocab_indicies_source,
				file_info.h_input_vocab_indicies_target,file_info.h_output_vocab_indicies_target,
				file_info.current_source_length,file_info.current_target_length,
				file_info.h_input_vocab_indicies_source_Wgrad,file_info.h_input_vocab_indicies_target_Wgrad,
				file_info.len_source_Wgrad,file_info.len_target_Wgrad,file_info.h_sampled_indices,
				file_info.len_unique_words_trunc_softmax);

			// hipProfilerStop();
			// return 0;

			end_minibatch = std::chrono::system_clock::now();
			elapsed_seconds = end_minibatch-begin_minibatch;

			total_batch_time_SPEED+= elapsed_seconds.count();
			total_words_batch_SPEED+=file_info.words_in_minibatch;

			if(curr_batch_num_SPEED>=thres_batch_num_SPEED) {
				std::cout << "Batched Minibatch time: " << total_batch_time_SPEED/60.0 << " minutes\n";
				std::cout << "Batched Words in minibatch: " << total_words_batch_SPEED << "\n";
				std::cout << "Batched Throughput: " << (total_words_batch_SPEED)/(total_batch_time_SPEED) << " words per second\n";
				std::cout << total_words << " out of " << params.train_total_words << " epoch: " << current_epoch <<  "\n\n";
				if(params.HPC_output) {
					HPC_output << "Batched Minibatch time: " << total_batch_time_SPEED/60.0 << " minutes\n";
					HPC_output << "Batched Words in minibatch: " << total_words_batch_SPEED << "\n";
					HPC_output << "Batched Throughput: " << (total_words_batch_SPEED)/(total_batch_time_SPEED) << " words per second\n";
					HPC_output << total_words << " out of " << params.train_total_words << " epoch: " << current_epoch <<  "\n\n";
					HPC_output.flush();
				}
				total_words_batch_SPEED = 0;
				total_batch_time_SPEED = 0;
				curr_batch_num_SPEED = 0;

			}
			curr_batch_num_SPEED++;
			total_words += file_info.words_in_minibatch;

			//stuff for google learning rate
			if(params.google_learning_rate && current_epoch>=params.epoch_to_start_halving && total_words>=params.half_way_count &&
				learning_rate_flag) {
					temp_learning_rate = temp_learning_rate/2;
					std::cout << "New Learning Rate: " << temp_learning_rate << "\n";
					model.update_learning_rate(temp_learning_rate);
					learning_rate_flag = false;
					if(params.HPC_output) {
						HPC_output << "New Learning Rate: " << temp_learning_rate << "\n";
						HPC_output.flush();
					}
			}

			//stuff for perplexity based learning schedule
			if(params.learning_rate_schedule && total_words>=params.half_way_count &&learning_rate_flag) {
				learning_rate_flag = false;
				double new_perplexity = model.get_perplexity(params.test_file_name,params.minibatch_size,params.test_num_lines_in_file,params.longest_sent,
					params.source_vocab_size,params.target_vocab_size,HPC_output,false,params.test_total_words,params.HPC_output,false,"");
				std::cout << "Old dev set Perplexity: " << old_perplexity << "\n";
				std::cout << "New dev set Perplexity: " << new_perplexity << "\n";
				if(params.HPC_output) {
					HPC_output << "Old dev set Perplexity: " << old_perplexity << "\n";
					HPC_output << "New dev set Perplexity: " << new_perplexity << "\n";
					HPC_output.flush();
				}
				if ( (new_perplexity + params.margin >= old_perplexity) && current_epoch!=1) {
					temp_learning_rate = temp_learning_rate*params.decrease_factor;
					model.update_learning_rate(temp_learning_rate);
					std::cout << "New learning rate:" << temp_learning_rate <<"\n\n";
					if(params.HPC_output) {
						HPC_output << "New learning rate:" << temp_learning_rate <<"\n\n";
						HPC_output.flush();
					}
				}
				//perplexity is better so output the best model file
				if(params.best_model && params.best_model_perp > new_perplexity) {
					std::cout << "Now outputting the new best model\n";
					model.dump_best_model(params.best_model_file_name,params.output_weight_file);
					if(params.HPC_output) {
							HPC_output << "Now outputting the new best model\n";
							HPC_output.flush();
					}
					params.best_model_perp = new_perplexity;
				}
			
				old_perplexity = new_perplexity;
			}

			if(!success) {
				current_epoch+=1;
				//stuff for google learning rate schedule
				if(params.google_learning_rate && current_epoch>=params.epoch_to_start_halving) {
					temp_learning_rate = temp_learning_rate/2;
					std::cout << "New learning rate:" << temp_learning_rate <<"\n\n";
					model.update_learning_rate(temp_learning_rate);
					learning_rate_flag = true;
					if(params.HPC_output) {
						HPC_output << "New learning rate:" << temp_learning_rate <<"\n\n";
						HPC_output.flush();
					}
				}

				double new_perplexity;
				if(params.google_learning_rate || params.learning_rate_schedule) {
					new_perplexity = model.get_perplexity(params.test_file_name,params.minibatch_size,params.test_num_lines_in_file,params.longest_sent,
						params.source_vocab_size,params.target_vocab_size,HPC_output,false,params.test_total_words,params.HPC_output,false,"");
				}
				//stuff for perplexity based learning schedule
				if(params.learning_rate_schedule) {
					std::cout << "Old dev set Perplexity: " << old_perplexity << "\n";
					std::cout << "New dev set Perplexity: " << new_perplexity << "\n";
					if(params.HPC_output) {
						HPC_output << "Old dev set Perplexity: " << old_perplexity << "\n";
						HPC_output << "New dev set Perplexity: " << new_perplexity << "\n";
						HPC_output.flush();
					}
					if ( (new_perplexity + params.margin >= old_perplexity) && current_epoch!=1) {
						if(params.HPC_output) {
							HPC_output << "New learning rate:" << temp_learning_rate <<"\n\n";
							HPC_output.flush();
						}
						temp_learning_rate = temp_learning_rate*params.decrease_factor;
						model.update_learning_rate(temp_learning_rate);
						std::cout << "New learning rate:" << temp_learning_rate <<"\n\n";
					}
					

					//perplexity is better so output the best model file
					if(params.best_model && params.best_model_perp > new_perplexity) {
						std::cout << "Now outputting the new best model\n";
						model.dump_best_model(params.best_model_file_name,params.output_weight_file);
						if(params.HPC_output) {
								HPC_output << "Now outputting the new best model\n";
								HPC_output.flush();
						}
						params.best_model_perp = new_perplexity;
					}


					learning_rate_flag = true;
					old_perplexity = new_perplexity;
				}

				if(params.train_perplexity) {
					std::cout << "PData on train set:"  << model.train_perplexity << "\n";
					std::cout << "Total target words: " << file_info.total_target_words << "\n";
					std::cout << "Training set perplexity: " << std::pow(2,-1*model.train_perplexity/file_info.total_target_words) << "\n";
					if(params.HPC_output) {
						HPC_output << "Training set perplexity: " << std::pow(2,-1*model.train_perplexity/file_info.total_target_words) << "\n";
						HPC_output.flush();
					}
					model.train_perplexity = 0;
				}

				total_words=0;
				if(current_epoch <= params.num_epochs) {
					std::cout << "-----------------------------------"  << std::endl;
					std::cout << "Starting epoch " << current_epoch << std::endl;
					std::cout << "-----------------------------------"  << std::endl;
					if(params.HPC_output) {
						HPC_output << "-----------------------------------"  << std::endl;
						HPC_output << "Starting epoch " << current_epoch << std::endl;
						HPC_output << "-----------------------------------"  << std::endl;
						HPC_output.flush();
					}
				}
			}
			hipDeviceSynchronize();
		}	
		//Now that training is done, dump the weights
		hipDeviceSynchronize();
		model.dump_weights();
	}


	/////////////////////////////////Get perplexity on test set////////////////////////////////
	if(params.test) {
		model.get_perplexity(params.test_file_name,params.minibatch_size,params.test_num_lines_in_file,params.longest_sent,
			params.source_vocab_size,params.target_vocab_size,HPC_output,true,params.test_total_words,params.HPC_output,true,params.output_force_decode);
	}

	if(params.LM && params.stochastic_generation) {
		model.stoicastic_generation(params.sg_length,params.sg_output_file_temp,params.temperature);
		input_file_prep input_helper;
		input_helper.unint_file(params.input_weight_file,params.sg_output_file_temp,params.sg_output_file,true,false);
	}


	///////////////////////////////////////////decode the model////////////////////////////////////////////
	if(params.decode) {
		std::cout << "-----------------Starting Decoding----------------\n";
		begin_decoding = std::chrono::system_clock::now();
		model.beam_decoder(params.beam_size,params.decode_file_name,
			params.input_weight_file,params.decode_num_lines_in_file,params.source_vocab_size,
			params.target_vocab_size,params.longest_sent,params.LSTM_size,params.penalty,
			params.decoder_output_file,params.min_decoding_ratio,params.max_decoding_ratio,params.softmax_scaled,
			params.num_hypotheses,params.print_score,params.dump_LSTM,params.LSTM_dump_file);
		end_decoding = std::chrono::system_clock::now();
		std::chrono::duration<double> elapsed_seconds = end_decoding-begin_decoding;
		std::cout << "Decoding time: " << elapsed_seconds.count()/60.0 << " minutes\n";

		//now unintegerize the file
		input_file_prep input_helper;
		input_helper.unint_file(params.input_weight_file,params.decoder_output_file,params.decoder_final_file,false,true);
	}



	//remove the temp directory created
	if(params.unique_dir!="NULL") {
		boost::filesystem::path temp_path(params.unique_dir);
		boost::filesystem::remove_all(temp_path);
	}

	//Compute the final runtime
	end_total = std::chrono::system_clock::now();
	elapsed_seconds = end_total-start_total;
    std::cout << "\n\n\n";
    std::cout << "Total Program Runtime: " << elapsed_seconds.count()/60.0 << " minutes" << std::endl;
}
